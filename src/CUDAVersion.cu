#include "hip/hip_runtime.h"
#include "../include/CUDAVersion.cuh"

CUDAVersion::CUDAVersion(std::string graphFilename, unsigned vertexesNumber){
	this->init(graphFilename, vertexesNumber);
	tab_sizes = new int[vertexesNumber];

	for (int i = 0; i < vertexesNumber; i++)	// ujemne wagi
		for (int j = 0; j < vertexesNumber; j++)
			matrix[i][j] = -matrix[i][j];

	if (true) {
		int device;
		hipGetDevice(&device);
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, device);
		threadsNumber = properties.maxThreadsDim[0];
		blocksNumber = std::min(properties.maxGridSize[0], ((int)vertexesNumber + threadsNumber - 1) / threadsNumber);
	} else {
		blocksNumber = 1;
		threadsNumber = 32;
	}

	fillAdjacencyTable();
}

/*CUDAVersion::~CUDAVersion() {
	delete
}*/

void CUDAVersion::fillAdjacencyTable() {

	for (int i = 0; i < vertexesNumber; i++) {
		tab_sizes[i] = 0;
		for (int j = 0; j < vertexesNumber; j++) {
			if (matrix[i][j] <= 0) {
				stab[i * vertexesNumber + j].first = j;
				stab[i * vertexesNumber + j].second = matrix[i][j];
				tab_sizes[i]++;
				//tab_end[i].push_back(j);
			}
		}
	}
	 /*
	int i, j, size;
	adjacency_table = new edges[vertexesNumber];
	for (i = 0; i < vertexesNumber; i++) {
		for (j = i; j < vertexesNumber; j++) {
			if (matrix[i][j] != 0) {
				adjacency_table[i].push_back(new std::pair<int, int>(j, matrix[i][j]));
			}
		}
	}

	//std::pair<int, int> tablica[vertexes][vertexes]

	tab = new std::pair<int, int>**[vertexesNumber];
	for (i = 0; i < vertexesNumber; i++) {
		tab_sizes[i] = adjacency_table[i].size();
		//std::cout << "size: " << tab_sizes[i] << std::endl;

		if (size == 0) {
			tab[i] = nullptr;// new std::pair<int, int>*[1];
							 //tab[i][0] = new std::pair<int, int>(i, 0);
			continue;
		}
		else {
			tab[i] = new std::pair<int, int>*[tab_sizes[i]];
		}

		for (j = 0; j < tab_sizes[i]; j++) {
			tab[i][j] = new std::pair<int, int>(adjacency_table[i][j]->first, adjacency_table[i][j]->second);
		}
	}
	*/
}

AbstractGraph::path* CUDAVersion::getCriticalPath(unsigned vertexStart) {
	path* res = new path();

	hipDeviceReset();

	std::pair<std::vector<int>, std::vector<unsigned>> pair;
	bf(vertexStart, &pair);
	int intIndex = std::min_element(pair.first.begin(), pair.first.end()) - pair.first.begin();
	res->pathLength = -pair.first[intIndex];

	return res;
}

AbstractGraph::path * CUDAVersion::getCriticalPath() {
	return getCriticalPath(0);
}

__global__ void kernel_old(unsigned vertexesNumber, int* matrix, int* distance) {
	int globalIndex = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;

	if (globalIndex >= vertexesNumber)
		return;

	for (int i = 0; i < vertexesNumber; i++) {
		for (int j = globalIndex; j < vertexesNumber; j += offset) {
			if (matrix[i * vertexesNumber + j] != 0) {
				if (distance[j] > distance[i] + matrix[i * vertexesNumber + j]) {
					distance[j] = distance[i] + matrix[i * vertexesNumber + j];
				}
			}
		}
	}
}

__global__ void kernelNew_old(unsigned vertexesNumber, int* matrix, int* distance) {
	int globalIndex = blockDim.x * blockIdx.x + threadIdx.x;
	int sum, weight;

	if (globalIndex >= vertexesNumber)
		return;
	
	for (int j = 0; j < vertexesNumber; j++) {
		weight = matrix[globalIndex * vertexesNumber + j];
		if (weight != 0) {
			sum = distance[globalIndex] + weight;
			if (distance[j] > weight) {
				atomicMin(&(distance[j]), sum);
			}
		}
	}
}

// aktualizacja distance po przejsciu edges
// wi�cej niz jedna wspolrzedna na kernel

void CUDAVersion::bellmanFord_old(unsigned row, std::pair<std::vector<int>, std::vector<unsigned>>* pair) {
	int* distance = new int[vertexesNumber];
	int* cuda_distance;
	int* cuda_matrix;
	std::vector<unsigned> predecessor;

	dim3 blocks(blocksNumber);
	dim3 threads(threadsNumber);

	hipMalloc(&cuda_matrix, sizeof(int) * vertexesNumber * vertexesNumber);
	hipMalloc(&cuda_distance, sizeof(int) * vertexesNumber);

	for (int i = 0; i < vertexesNumber; i++) {
		distance[i] = INT_MAX;
	}

	distance[row] = 0;
	hipMemcpy(cuda_distance, distance, sizeof(int) * vertexesNumber, hipMemcpyHostToDevice);
	//hipMemcpy(cuda_distance, distance, sizeof(long) * vertexesNumber, hipMemcpyHostToDevice);
	hipMemcpy(cuda_matrix, linear_matrix, sizeof(int) * vertexesNumber * vertexesNumber, hipMemcpyHostToDevice);

	//void* args[] = { &vertexesNumber, cuda_matrix, cuda_distance};
	//hipLaunchKernel((const void*)&kernel, blocks, threads, args);
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	//kernel<<<blocks, threads>>>(vertexesNumber, cuda_matrix, cuda_distance);
	kernelNew_old<<<blocks, threads>>>(vertexesNumber, cuda_matrix, cuda_distance);
	//kernelNew << <1, vertexesNumber>> >(vertexesNumber, cuda_matrix, cuda_distance);

	hipDeviceSynchronize();

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&miliseconds, start, stop);

	hipMemcpy(distance, cuda_distance, sizeof(int) * vertexesNumber, hipMemcpyDeviceToHost);

	hipFree(cuda_matrix);
	hipFree(cuda_distance);

	//for (int k = 0; k < vertexesNumber; k++)
	//	std::cout << distance[k] << std::endl;

	pair->first = std::vector<int>(distance, distance + vertexesNumber);

	//for(int w = 0; w < vertexesNumber; w++)
	//	std::cout << pair->first[w] << std::endl;

	pair->second = predecessor;
}

// ---------------------------------------------------------------------------------------------

__global__ void relax_old(unsigned vertexesNumber, unsigned edgesAmount, int edgeStart, int* cuda_matrix, int* cuda_distance) {
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int sum, weight;

	printf("id = %d, blockIdx = %d, threardIdx = %d\n", id, blockIdx.x, threadIdx.x);

	if (id >= edgesAmount) return;
	printf("dupa2\n");


		weight = cuda_matrix[edgeStart * vertexesNumber + id];
			printf("dupa3\n");
		if (weight != 0) {
			printf("dupa4\n");

			sum = cuda_distance[edgeStart] + weight;
			printf("dupa5\n");

			if (cuda_distance[id] > weight) {
				printf("dupa6\n");

				atomicMin(&(cuda_distance[id]), sum);
				printf("dupa7\n");

			}
		}
	
}

void CUDAVersion::bf_old(unsigned row, std::pair<std::vector<int>, std::vector<unsigned>>* pair) {
	int* distance = new int[vertexesNumber];
	int* return_distance = new int[vertexesNumber];

	int* cuda_distance;
	int* cuda_matrix;

	std::vector<unsigned> predecessor;

	dim3 blocks(blocksNumber);
	dim3 threads(threadsNumber);

	hipMalloc(&cuda_distance, sizeof(int) * vertexesNumber);
	hipMalloc(&cuda_matrix, sizeof(int) * vertexesNumber * vertexesNumber);

	for (int i = 0; i < vertexesNumber; i++) {
		distance[i] = INT_MAX;
	}

	distance[row] = 0;
	
	hipMemcpy(cuda_distance, distance, sizeof(int) * vertexesNumber, hipMemcpyHostToDevice);
	hipMemcpy(cuda_matrix, linear_matrix, sizeof(int) * vertexesNumber * vertexesNumber, hipMemcpyHostToDevice);

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	int b, t, edgesAmount;

	for (int i = 0; i < vertexesNumber; i++) { // wywolujemy tyle watkow ile mamy par

		edgesAmount = tab_sizes[i];
		//std::cout << "jebac4: " << edgesAmount << std::endl;

		if (edgesAmount == 0) continue;

		b = (edgesAmount / 24) + 1; // liczba blokow
		if (b == 1) t = edgesAmount; else t = 24;

		//std::cout << "b: " << b << "t: " << t << std::endl;
		relax_old <<<b, t>>> (vertexesNumber, edgesAmount, i, cuda_matrix, cuda_distance);

	}

	hipDeviceSynchronize();

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&miliseconds, start, stop);

	hipMemcpy(return_distance, cuda_distance, sizeof(int) * vertexesNumber, hipMemcpyDeviceToHost);

	hipFree(cuda_distance);
	hipFree(cuda_matrix);

	//for (int k = 0; k < vertexesNumber; k++)
	//std::cout << return_distance[k] << std::endl;

	pair->first = std::vector<int>(return_distance, return_distance + vertexesNumber);
	pair->second = predecessor;
}

__global__ void initNodeWeight(unsigned row, unsigned vertexesNumber, int* cuda_distance) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id >= vertexesNumber) return;

	cuda_distance[id] = INT_MAX;
	if (id == row)
		cuda_distance[row] = 0;

}

__global__ void relax(unsigned edgesAmount, int edgeStart, std::pair<int, int>*** cuda_adjacency_table, int* cuda_distance) {
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	//if (threadIdx.x == 10 && blockIdx.x == 0)
	printf("id = %d, blockIdx = %d, threardIdx = %d\n", id, blockIdx.x, threadIdx.x);


	if (id >= edgesAmount) return;

	//if (threadIdx.x == 10)
		printf("dupa2\n");

	if (cuda_adjacency_table[edgeStart] == nullptr) return;
	//if (threadIdx.x == 10)
		printf("dupa3\n");

	int endVertex = cuda_adjacency_table[edgeStart][id]->first;
	//if (threadIdx.x == 10)
		printf("dupa4\n");

	int weight = cuda_adjacency_table[edgeStart][id]->second;

	//if (threadIdx.x == 10)
		printf("dupa5\n");

	if (cuda_distance[endVertex] > cuda_distance[edgeStart] + weight) {
		//if (threadIdx.x == 10)
			printf("dupa6\n");

		atomicMin((cuda_distance + sizeof(int) * endVertex), (cuda_distance[edgeStart] + weight));
	}
}

__global__ void greg(unsigned vertexesNumber, unsigned edgesAmount, int edgeStart, std::pair<int, int>* cuda_stab, int* cuda_distance) {
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	//if (threadIdx.x == 10 && blockIdx.x == 0)
	//printf("id = %d, blockIdx = %d, threardIdx = %d\n", id, blockIdx.x, threadIdx.x);

	if (id >= edgesAmount) return;

	//if (threadIdx.x == 10)
	//printf("dupa2\n");

	int endVertex = cuda_stab[edgeStart * vertexesNumber + id].first;
	//if (threadIdx.x == 10)
	//printf("dupa4\n");

	int weight = cuda_stab[edgeStart * vertexesNumber + id].second;

	if (weight >= 0) return;

	//if (threadIdx.x == 10)
	//printf("dupa5\n");

	if (cuda_distance[endVertex] > cuda_distance[edgeStart] + weight) {
		//if (threadIdx.x == 10)
		//printf("dupa6\n");
		//cuda_distance[endVertex] = cuda_distance[edgeStart] + weight;
		//atomicMin((cuda_distance + sizeof(int) * endVertex), (cuda_distance[edgeStart] + weight));
		atomicMin(&(cuda_distance[endVertex]), (cuda_distance[edgeStart] + weight));

	}
}


void CUDAVersion::bf(unsigned row, std::pair<std::vector<int>, std::vector<unsigned>>* pair) {
	int* distance = new int[vertexesNumber];
	int* return_distance = new int[vertexesNumber];

	int* cuda_distance;
	//int* cuda_matrix;

	std::pair<int, int>* cuda_stab;
	std::vector<unsigned> predecessor;

	dim3 blocks(blocksNumber);
	dim3 threads(threadsNumber);

	hipMalloc(&cuda_stab, sizeof(std::pair<int,int>) * vertexesNumber * vertexesNumber);
	hipMalloc(&cuda_distance, sizeof(int) * vertexesNumber);
	//hipMalloc(&cuda_matrix, sizeof(int) * vertexesNumber * vertexesNumber);


	hipMemcpy(cuda_stab, stab, sizeof(std::pair<int, int>) * vertexesNumber * vertexesNumber, hipMemcpyHostToDevice);
	//hipMemcpy(cuda_matrix, linear_matrix, sizeof(int) * vertexesNumber * vertexesNumber, hipMemcpyHostToDevice);

	for (int i = 0; i < vertexesNumber; i++) {
		distance[i] = INT_MAX;
	}

	distance[row] = 0;
	hipMemcpy(cuda_distance, distance, sizeof(int) * vertexesNumber, hipMemcpyHostToDevice);

	//initNodeWeight <<<blocksNumber, threadsNumber>>>(row, vertexesNumber, cuda_distance);

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	int b, t, edgesAmount;

	for (int i = 0; i < vertexesNumber; i++) { // wywolujemy tyle watkow ile mamy par
		
		edgesAmount = tab_sizes[i];
		//std::cout << "jebac4: " << edgesAmount << std::endl;

		if (edgesAmount == 0) {
			continue;
		}

/*		if (edgesAmount != tab_end[i].size()) {
			std::cout << "xxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxx" << std::endl;
		}*/
		
		b = (edgesAmount / 24) + 1; // liczba blokow
		if (b == 1)
			t = edgesAmount;
		else 
			t = 24;

		//if( i > 1950)
		//std::cout << "edgesAmount:\t" << edgesAmount << "\tb:\t" << b << "\tt:\t" << t << std::endl;
		greg <<<b, t>>> (vertexesNumber, edgesAmount, i, cuda_stab, cuda_distance);
		//relax_old <<<b, t>>> (vertexesNumber, edgesAmount, i, cuda_matrix, cuda_distance);

	}

	hipDeviceSynchronize();

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&miliseconds, start, stop);

	hipMemcpy(return_distance, cuda_distance, sizeof(int) * vertexesNumber, hipMemcpyDeviceToHost);

	hipFree(cuda_stab);
	hipFree(cuda_distance);
	//hipFree(cuda_matrix);

	/*int temp = INT_MAX;
	for (int k = 0; k < vertexesNumber; k++) {
		if (temp > return_distance[k]) {
			temp = return_distance[k];
			std::cout << return_distance[k] << std::endl;
		}
	}*/

	pair->first = std::vector<int>(return_distance, return_distance + vertexesNumber);
	pair->second = predecessor;
}