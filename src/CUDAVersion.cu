#include "hip/hip_runtime.h"
#include "../include/CUDAVersion.cuh"

CUDAVersion::CUDAVersion(std::string graphFilename, unsigned vertexesNumber){
	this->init(graphFilename, vertexesNumber);

	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	threadsNumber = properties.maxThreadsDim[0];
	blocksNumber = std::min(properties.maxGridSize[0], ((int)vertexesNumber + threadsNumber - 1) / threadsNumber);
}

AbstractGraph::path* CUDAVersion::getCriticalPath(unsigned vertexStart) {
	for (int i = 0; i < vertexesNumber; i++)	// ujemne wagi
		for (int j = 0; j < vertexesNumber; j++)
			linear_matrix[i * vertexesNumber + j] = -linear_matrix[i * vertexesNumber + j];

	path* res = new path();

	hipDeviceReset();

	std::pair<std::vector<long>, std::vector<unsigned>> pair;
	bellmanFord(vertexStart, &pair);
	int intIndex = std::min_element(pair.first.begin(), pair.first.end()) - pair.first.begin();
	res->pathLength = -pair.first[intIndex];

	return res;
}

AbstractGraph::path * CUDAVersion::getCriticalPath() {
	return getCriticalPath(0);
}

__global__ void kernel(unsigned vertexesNumber, long* matrix, long* distance) {
	int globalIndex = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;

	if (globalIndex >= vertexesNumber)
		return;

	for (int i = 0; i < vertexesNumber; i++) {
		for (int j = globalIndex; j < vertexesNumber; j += offset) {
			if (matrix[i * vertexesNumber + j] != 0) {
				if (distance[j] > distance[i] + matrix[i * vertexesNumber + j]) {
					distance[j] = distance[i] + matrix[i * vertexesNumber + j];
				}
			}
		}
	}
}

void CUDAVersion::bellmanFord(unsigned row, std::pair<std::vector<long>, std::vector<unsigned>>* pair) {
	long* distance = new long[vertexesNumber];
	long* cuda_distance;
	long* cuda_matrix;
	std::vector<unsigned> predecessor;

	dim3 blocks(blocksNumber);
	dim3 threads(threadsNumber);

	hipMalloc(&cuda_matrix, sizeof(long) * vertexesNumber * vertexesNumber);
	hipMalloc(&cuda_distance, sizeof(long) * vertexesNumber);

	for (int i = 0; i < vertexesNumber; i++) {
		distance[i] = LONG_MAX;
	}

	distance[row] = 0;
	hipMemcpy(cuda_distance, distance, sizeof(long) * vertexesNumber, hipMemcpyHostToDevice);
	hipMemcpy(cuda_matrix, linear_matrix, sizeof(long) * vertexesNumber * vertexesNumber, hipMemcpyHostToDevice);

	//void* args[] = { &vertexesNumber, cuda_matrix, cuda_distance};
	//hipLaunchKernel((const void*)&kernel, blocks, threads, args);

	kernel<<<blocks, threads>>>(vertexesNumber, cuda_matrix, cuda_distance);
	//kernel <<<blocksNumber, threadsNumber>>>(vertexesNumber, cuda_matrix, cuda_distance);

	hipDeviceSynchronize();

	hipMemcpy(distance, cuda_distance, sizeof(long) * vertexesNumber, hipMemcpyDeviceToHost);

	hipFree(cuda_matrix);
	hipFree(cuda_distance);

	//for (int k = 0; k < vertexesNumber; k++)
	//	std::cout << distance[k] << std::endl;

	pair->first = std::vector<long>(distance, distance + vertexesNumber);

	//for(int w = 0; w < vertexesNumber; w++)
	//	std::cout << pair->first[w] << std::endl;

	pair->second = predecessor;
}