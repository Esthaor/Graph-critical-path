#include "hip/hip_runtime.h"
#include "../include/CudaVersion3.cuh"

CudaVersion3::CudaVersion3(std::string graphFilename, unsigned vertexesNumber) {
	this->init(graphFilename, vertexesNumber);
	adjacency_table = new edges[vertexesNumber];
	tab_sizes = new int[vertexesNumber];

	for (int i = 0; i < vertexesNumber; i++)	// ujemne wagi
		for (int j = 0; j < vertexesNumber; j++)
			matrix[i][j] = -matrix[i][j];

	fillAdjacencyTable();
}

void CudaVersion3::fillAdjacencyTable() {
	int i, j, size;
	for (i = 0; i < vertexesNumber; i++) {
		for (j = i; j < vertexesNumber; j++) {
			if (matrix[i][j] != 0) {
				adjacency_table[i].push_back(new std::pair<int, int>(j, matrix[i][j]));
			}
		}
	}

	tab = new std::pair<int, int>*[vertexesNumber];
	for (i = 0; i < vertexesNumber; i++) {
		tab_sizes[i] = adjacency_table[i].size();

		if (size == 0) {
			tab[i] = nullptr;
			continue;
		}

		tab[i] = new std::pair<int, int>[tab_sizes[i]];


		for (j = 0; j < tab_sizes[i]; j++) {
			tab[i][j] = std::pair<int, int>(adjacency_table[i][j]->first, adjacency_table[i][j]->second);
		}
	}
}

AbstractGraph::path* CudaVersion3::getCriticalPath(unsigned vertexStart) {
	path* res = new path();

	hipDeviceReset();

	std::pair<std::vector<int>, std::vector<unsigned>> pair;
	bf(vertexStart, &pair);
	int intIndex = std::min_element(pair.first.begin(), pair.first.end()) - pair.first.begin();
	res->pathLength = -pair.first[intIndex];

	return res;
}

AbstractGraph::path * CudaVersion3::getCriticalPath() {
	return getCriticalPath(0);
}

__global__ void initNodeWeight_adj(unsigned row, unsigned vertexesNumber, int* cuda_distance) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id >= vertexesNumber) return;

	cuda_distance[id] = INT_MAX;
	if (id == row)
		cuda_distance[row] = 0;

}

__global__ void relax_adj(unsigned vertexesNumber, unsigned edgesAmount, int edgeStart, std::pair<int, int>** cuda_adjacency_table, int* cuda_distance) {
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	//if (threadIdx.x == 10 && blockIdx.x == 0)
	printf("id = %d, blockIdx = %d, threardIdx = %d\n", id, blockIdx.x, threadIdx.x);


	if (id >= edgesAmount) return;

	//if (threadIdx.x == 10)
	printf("dupa2\n");

	if (cuda_adjacency_table[edgeStart] == nullptr) return;
	//if (threadIdx.x == 10)
	printf("dupa3\n");

	int endVertex = cuda_adjacency_table[edgeStart][id].first;
	//if (threadIdx.x == 10)
	printf("dupa4\n");

	int weight = cuda_adjacency_table[edgeStart][id].second;

	//if (threadIdx.x == 10)
	printf("dupa5\n");

	if (cuda_distance[endVertex] > cuda_distance[edgeStart] + weight) {
		//if (threadIdx.x == 10)
		printf("dupa6\n");

		atomicMin((cuda_distance + sizeof(int) * endVertex), (cuda_distance[edgeStart] + weight));
	}
}

void CudaVersion3::bf(unsigned row, std::pair<std::vector<int>, std::vector<unsigned>>* pair) {
	std::cout << "bf1" << std::endl;

	int* distance = new int[vertexesNumber];
	int* return_distance = new int[vertexesNumber];

	int* cuda_distance;
	//int* cuda_matrix;

	std::pair<int, int>** cuda_adjacency;
	std::pair<int, int>** cuda_adjacency_to_copy = new std::pair<int, int>*[vertexesNumber];

	std::pair<int, int>* temp_tab_pair;
	//std::pair<int, int>* temp_pair;

	std::vector<unsigned> predecessor;

	dim3 blocks(blocksNumber);
	dim3 threads(threadsNumber);

	hipMalloc(&cuda_adjacency, sizeof(std::pair<int, int>*) * vertexesNumber);
	std::cout << "bf2" << std::endl;


	for (int m = 0; m < vertexesNumber; m++) {
		std::cout << "bf2-----------------------------------" << std::endl;

		hipMalloc(&temp_tab_pair, sizeof(std::pair<int, int>) * tab_sizes[m]);

		for (int n = 0; n < tab_sizes[m]; n++) {
			hipMemcpy(&temp_tab_pair[n], &tab[m][n], sizeof(std::pair<int, int>), hipMemcpyHostToDevice);
		}

		std::cout << "bf2###############################################################################################" << std::endl;


		cuda_adjacency_to_copy[m] = temp_tab_pair;
	}
		std::cout << "bf3" << std::endl;


	hipMalloc(&cuda_distance, sizeof(int) * vertexesNumber);

	hipMemcpy(cuda_adjacency, cuda_adjacency_to_copy, sizeof(std::pair<int, int>*) * vertexesNumber, hipMemcpyHostToDevice);
	std::cout << "bf4" << std::endl;

	for (int i = 0; i < vertexesNumber; i++) {
		distance[i] = INT_MAX;
	}

	distance[row] = 0;
	hipMemcpy(cuda_distance, distance, sizeof(int) * vertexesNumber, hipMemcpyHostToDevice);

	//initNodeWeight <<<blocksNumber, threadsNumber>>>(row, vertexesNumber, cuda_distance);

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	int b, t, edgesAmount;
	std::cout << "bf5" << std::endl;

	for (int i = 0; i < vertexesNumber; i++) { // wywolujemy tyle watkow ile mamy par

		edgesAmount = tab_sizes[i];
		//std::cout << "jebac4: " << edgesAmount << std::endl;

		if (edgesAmount == 0) {
			continue;
		}

		/*		if (edgesAmount != tab_end[i].size()) {
		std::cout << "xxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxxx" << std::endl;
		}*/

		b = (edgesAmount / 24) + 1; // liczba blokow
		if (b == 1)
			t = edgesAmount;
		else
			t = 24;
		std::cout << "bf6" << std::endl;

		//if( i > 1950)
		//std::cout << "edgesAmount:\t" << edgesAmount << "\tb:\t" << b << "\tt:\t" << t << std::endl;
		relax_adj << <b, t >> > (vertexesNumber, edgesAmount, i, cuda_adjacency, cuda_distance);
		//relax_old <<<b, t>>> (vertexesNumber, edgesAmount, i, cuda_matrix, cuda_distance);

	}

	hipDeviceSynchronize();

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&miliseconds, start, stop);

	hipMemcpy(return_distance, cuda_distance, sizeof(int) * vertexesNumber, hipMemcpyDeviceToHost);

	for (int h = 0; h < vertexesNumber; h++) {
		hipFree(cuda_adjacency[h]);
	}
	hipFree(cuda_adjacency);
	hipFree(cuda_distance);
	//hipFree(cuda_matrix);

	/*int temp = INT_MAX;
	for (int k = 0; k < vertexesNumber; k++) {
	if (temp > return_distance[k]) {
	temp = return_distance[k];
	std::cout << return_distance[k] << std::endl;
	}
	}*/

	pair->first = std::vector<int>(return_distance, return_distance + vertexesNumber);
	pair->second = predecessor;
}